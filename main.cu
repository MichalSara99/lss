﻿#include<iostream>
#include<string>

#include"unit_tests/sparse_solvers/lss_core_cuda_solver_t.hpp"
#include"unit_tests/ode_solvers/second_degree/lss_odes_2_degree_t.hpp"
#include"unit_tests/pde_solvers/1d/lss_advection_equation_t.hpp"
#include"unit_tests/pde_solvers/1d/lss_black_scholes_equation_t.hpp"
#include"unit_tests/pde_solvers/1d/lss_pure_heat_equation_t.hpp"
#include"unit_tests/pde_solvers/1d/lss_pure_wave_equation_t.hpp"
#include"unit_tests/pde_solvers/2d/lss_heston_equation_t.hpp"
#include"unit_tests/pde_solvers/3d/lss_hhw_equation_t.hpp"
#include"unit_tests/pde_solvers/2d/lss_sabr_equation_t.hpp"
#include"unit_tests/common/lss_xml_t.hpp"
#include"unit_tests/common/lss_container_3d_t.hpp"

int main()
{
     // ======================================================
     // =============== lss_container_3d_t ===================
     // ======================================================
     // basic_rcontainer_3d_test();
     // slice_rcontainer_3d_test(); 
     // basic_ccontainer_3d_test();
     // slice_ccontainer_3d_test();  
     // basic_lcontainer_3d_test();

     // ======================================================
     // =============== lss_odes_2_degree_t ==================
     // ======================================================
     // test_impl_simple_ode_dirichlet_bc_cuda_solver_device();
     // test_impl_simple_ode_dirichlet_neumann_bc_cuda_solver_device();
     // test_impl_simple_ode_dirichlet_robin_bc_cuda_solver_device();
     // test_impl_simple_ode_neumann_robin_bc_cuda_solver_device();
     // test_impl_simple_ode1_neumann_robin_bc_cuda_solver_device();

    // ======================================================
    // ============= lss_core_cuda_solver_t =================
    // ======================================================
    // test_device_sparse_qr();
    // test_host_sparse_qr_test();
    // test_dirichlet_bc_bvp_on_host();
    // test_dirichlet_bc_bvp_on_device();
    // test_robin_bc_bvp_on_host();
    // test_robin_bc_bvp_on_device();

    // ======================================================

    // ======================================================
    // ============= lss_core_cuda_solver_t =================
    // ======================================================
    // test_impl_simple_ode_dirichlet_bc_cuda_solver_device();
    // test_impl_simple_ode_dirichlet_neumann_bc_cuda_solver_device();
    // test_impl_simple_ode_dirichlet_robin_bc_cuda_solver_device();
    // test_impl_simple_ode_neumann_robin_bc_cuda_solver_device();
    // test_impl_simple_ode1_neumann_robin_bc_cuda_solver_device();

    // ======================================================


    // ======================================================
    // =========== lss_advection_equation_t =================
    // ======================================================
    // test_impl_adv_diff_equation_dirichlet_bc_cuda_solver_device_qr();
    // test_impl_adv_diff_equation_dirichlet_bc_sor_solver_device();
    // test_impl_adv_diff_equation_dirichlet_bc_sor_solver_host();
    // test_impl_adv_diff_equation_dirichlet_bc_cuda_solver_host_qr();
    // test_impl_adv_diff_equation_dirichlet_bc_double_sweep_solver();
    // test_impl_adv_diff_equation_dirichlet_bc_thomas_lu_solver();

    // ======================================================

    // ======================================================
    // =========== lss_black_scholes_equation_t =============
    // ======================================================
    // test_impl_black_scholes_equation_dirichlet_bc_cuda_solver_device_qr();
    // test_impl_black_scholes_equation_dirichlet_bc_sor_solver_device();
    // test_impl_black_scholes_equation_dirichlet_bc_sor_solver_host();
    // test_impl_black_scholes_equation_dirichlet_bc_double_sweep_solver();
    // test_impl_black_scholes_equation_dirichlet_bc_thomas_lu_solver();
    // test_impl_fwd_black_scholes_equation_dirichlet_bc_cuda_solver_device_qr();
    // test_impl_black_scholes_equation_dirichlet_bc_thomas_lu_solver_stepping();
    // test_expl_black_scholes_equation_dirichlet_bc_ade();
    // 
    // ======================================================

    // ======================================================
    // ================== lss_pure_heat_equation_t ==========
    // ======================================================
    // test_impl_pure_heat_equation_dirichlet_bc_cuda_solver_device_qr();
    // test_impl_pure_heat_equation_dirichlet_bc_sor_solver_device();
    // test_impl_pure_heat_equation_dirichlet_bc_sor_solver_host();
    // test_impl_pure_heat_equation_dirichlet_bc_cuda_solver_host_qr();
    // test_impl_pure_heat_equation_dirichlet_bc_double_sweep_solver();
    // test_impl_pure_heat_equation_dirichlet_bc_thomas_lu_solver();
    // test_impl_pure_heat_equation_neumann_bc_cuda_solver_device_qr();
    // test_impl_pure_heat_equation_neumann_bc_thomas_lu_solver();
    // test_impl_pure_heat_equation_neumann_bc_double_sweep_solver();
    // test_impl_pure_heat_equation_dirichlet_bc_cuda_solver_device_qr_stepping();
    // test_impl_pure_heat_equation_source_dirichlet_bc_cuda_solver_device();
    // test_impl_pure_heat_equation_source_dirichlet_bc_sor_solver_device();
    // 
    // test_expl_pure_heat_equation_dirichlet_bc_ade();
    // test_expl_pure_heat_equation_neumann_bc_euler();
    // test_expl_pure_heat_equation_dirichlet_bc_device();
    // ======================================================

    // ======================================================
    // =============== lss_pure_wave_equation_t =============
    // ======================================================
    // test_impl_pure_wave_equation_dirichlet_bc_cuda_solver_device_qr();
    // test_impl_pure_wave_equation_dirichlet_bc_cuda_solver_device_sor();
    // test_impl_pure_wave_equation_dirichlet_bc_cuda_solver_host_sor();
    // test_impl_pure_wave_equation_dirichlet_bc_solver_host_double_sweep();
    // test_impl_pure_wave_equation_dirichlet_bc_solver_host_lu();
    // test_impl_wave_equation_dirichlet_bc_solver_host_lu();
    // test_impl_damped_wave_equation_dirichlet_bc_solver_host_double_sweep();
    // test_impl_pure_wave_equation_neumann_bc_cuda_solver_device_qr();
    // test_expl_pure_wave_equation_dirichlet_bc_cuda_host_solver();
    // test_expl_pure_wave_equation_dirichlet_bc_cuda_device_solver();
    // 
    // ======================================================

    // ======================================================
    // =============== lss_heston_equation_t ================
    // ======================================================
    // test_impl_heston_equation_cuda_qr_solver();
    // test_impl_heston_equation_thomas_lu_solver();
    //
    // ======================================================

    // ======================================================
    // ================= lss_sabr_equation_t ================
    // ======================================================
    // test_impl_sabr_equation_double_sweep_solver();
    //
    // ======================================================

    // ======================================================
    // ================== lss_hhw_equation_t ================
    // ======================================================
    // test_impl_hhw_equation_cuda_qr_solver();
    // test_impl_hhw_equation_tlu_dss_solver();  
    // 
    // ======================================================

    // ======================================================
    // ====================== lss_xml_t =====================
    // ======================================================
    // test_impl_simple_ode_thomes_lu_qr_xml();
    // test_impl_bs_thomas_lu_crv_xml();
    // test_impl_bs_thomas_lu_srf_xml();
    // test_impl_ph_dirichlet_bvp_device_qr_srf_xml();
    // test_expl_ph_neumann_bvp_euler_srf_xml();
    // test_impl_adv_thomas_lu_srf_xml();
    // test_impl_pw_dirichlet_bvp_cuda_device_qr_srf_xml();
    // test_impl_w_dirichlet_bvp_host_lu_srf_xml();
    // test_impl_w_bvp_host_dss_srf_xml();
    // test_impl_pw_neumann_bvp_cuda_device_qr_srf_xml();
    // test_expl_pw_dirichlet_bvp_cuda_host_host_srf_xml();
    // test_impl_heston_cuda_qr_cn_dr_srf_xml();
    // test_impl_heston_thomas_lu_cn_srf_xml();
    // test_impl_sabr_double_sweep_cn_srf_xml();
    // test_impl_heston_thomas_lu_dr_cn_srf_xml();
    // test_impl_heston_dss_cs_cn_srf_xml();
    // test_impl_heston_thomas_lu_mcs_cn_srf_xml();
    // test_impl_heston_thomas_lu_hw_cn_srf_xml();
    // test_expl_heston_host_euler_srf_xml();
    // test_expl_heston_device_euler_srf_xml();
    // test_expl_sabr_host_euler_srf_xml();
    // test_expl_sabr_device_euler_srf_xml();
    
    //
    // ======================================================

    std::cin.get();
    std::cin.get();
    return 0;
}

